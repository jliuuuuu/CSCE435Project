#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>


int THREADS;
int BLOCKS;
int NUM_VALS;

int kernel_calls;

/* Define Caliper region names */
const char *data_init = "data_init";
const char *comm = "comm";
const char *comm_small = "comm_small";
const char *comm_large = "comm_large";
const char *comp = "comp";
const char *comp_small = "comp_small";
const char *comp_large = "comp_large";


void print_elapsed(clock_t start, clock_t stop)
{
    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
    return (float)rand() / (float)RAND_MAX;
}

void array_print(float *arr, int length)
{
    int i;
    for (i = 0; i < length; ++i)
    {
        printf("%1.3f ", arr[i]);
    }
    printf("\n");
}

void array_fill(float *arr, int length)
{
    srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i)
    {
        arr[i] = random_float();
    }
}

/*Used ChatGPT for this portion*/
// Function to swap two elements in an array
__device__ void swap(float* arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}

// Function to partition the data array around a pivot
__device__ int partition(float* data, int left, int right, int pivot) {
    while (left <= right) {
        while (data[left] < pivot)
            left++;
        while (data[right] > pivot)
            right--;

        if (left <= right) {
            swap(data, left, right);
            left++;
            right--;
        }
    }
    return left;
}

// Function to select the median of medians as a pivot
__device__ int selectPivot(float* data, int left, int right) {
    int numElements = right - left + 1;
    int numMedians = (numElements + NUM_SAMPLES_PER_BLOCK - 1) / NUM_SAMPLES_PER_BLOCK;
    int* medians = data + left;

    for (int i = 0; i < numMedians; i++) {
        int medianIndex = left + i * NUM_SAMPLES_PER_BLOCK;
        int median = medians[i] = data[medianIndex];
        for (int j = i - 1; j >= 0 && medians[j] > median; j--) {
            medians[j + 1] = medians[j];
            medians[j] = median;
        }
    }

    return medians[numMedians / 2];
}

// Function to sort each block
__global__ void sortBlocks(float* data, int n) {
    int tid = threadIdx.x;
    int blockId = blockIdx.x;
    int startIndex = blockId * n;
    int endIndex = startIndex + n - 1;

    int pivot = selectPivot(data, startIndex, endIndex);
    int partitionIndex = partition(data, startIndex, endIndex, pivot);

    // Sort the data for this block using quicksort
    if (startIndex < partitionIndex - 1) {
        CALI_MARK_BEGIN(comp_small);
        sortBlocks(data, partitionIndex - startIndex);
        CALI_MARK_END(comp_small);
    }
    if (partitionIndex < endIndex) {
        CALI_MARK_BEGIN(comp_small);
        sortBlocks(data + partitionIndex, endIndex - partitionIndex + 1);
        CALI_MARK_END(comp_small);
    }
}

// Main function to perform Sample Sort 
void sample_sort(float* data, int n) {
    CALI_MARK_BEGIN(comp_large);
    sortBlocks<<<BLOCKS, THREADS>>>(data, n);
    CALI_MARK_END(comp_large);
}

/*End of ChatGPT portion*/

int main(int argc, char *argv[])
{
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    clock_t start, stop;

    float *values = (float *)malloc(NUM_VALS * sizeof(float));
    CALI_MARK_BEGIN(data_init);
    array_fill(values, NUM_VALS);
    CALI_MARK_END(data_init);

    start = clock();
    CALI_MARK_BEGIN(comp);
    sample_sort(values, NUM_VALS); /* Inplace */
    CALI_MARK_END(comp);
    stop = clock();

    print_elapsed(start, stop);


    adiak::init(NULL);
    adiak::launchdate();                                             // launch date of the job
    adiak::libraries();                                              // Libraries used
    adiak::cmdline();                                                // Command line used to launch the job
    adiak::clustername();                                            // Name of the cluster
    adiak::value("Algorithm", "SampleSort");                         // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA");                         // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float");                               // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float));                   // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS);                            // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random");                             // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    //adiak::value("num_procs", num_procs);                            // The number of processors (MPI ranks)
    adiak::value("num_threads", THREADS);                        // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS);                          // The number of CUDA blocks
    adiak::value("group_num", 21);                                   // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Online and AI (ChatGPT)") // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").


    // Flush Caliper output before finalizing MPI
    mgr.stop();
    mgr.flush();
}